#include <hip/hip_runtime.h>
#include "../include/core.cuh"

/*
 * Get the number of GPU device(s) on the machine
 */

inline int VLMO_get_device_num(const bool verbose=false) {

    int num_devices;
    cudaErrChk (hipGetDeviceCount (&num_devices));

    if (verbose == true) {
        printf("\n=================================================\n");
        printf("The number of device(s) : %d\n", num_devices);
        printf("=================================================\n\n");
    }

    return num_devices;

}



/*
 * Get properties of certain GPU device [device_id]
 */
hipDeviceProp_t VLMO_get_device_properties(const int device_id, size_t& free, size_t& total, const bool verbose=false) {

    hipDeviceProp_t prop;
    cudaErrChk ( hipGetDeviceProperties (&prop, device_id) );
    hipMemGetInfo (&free, &total);

    if (verbose == true) {
        printf ("Device Number: %d\n", device_id);
        printf ("  Device name: %s\n", prop.name);
        printf ("  Device compute capability: %d.%d\n", prop.major, prop.minor);
        printf ("  Number of SM(s): %d\n", prop.multiProcessorCount);
        printf ("  Memory Clock Rate (GHz): %.2f\n",
               ((float)prop.memoryClockRate)/1.0e6);
        printf ("  Memory Bus Width (bits): %d\n",
               prop.memoryBusWidth);
        printf ("  Peak Memory Bandwidth (GB/s): %f\n",
               2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);

        printf ("\n[Kernel size]\n");
        printf ("  Maximum size of a grid [%d, %d, %d]\n"
                , prop.maxGridSize[0], prop.maxGridSize[0], prop.maxGridSize[0]);
        printf ("  Maximum size of a block [%d]\n"
                , prop.maxThreadsPerBlock);
        printf ("\n[Global mem]\n");
        printf ("  Global memory size :%dKB\n", (int)(total/1.0e3));
        printf ("  Free memory size :%dKB\n", (int)(free/1.0e3));
        printf ("\n[Shared mem]\n");
        printf ("  Shared memory size per block :%dKB\n", (int)(prop.sharedMemPerBlock/1.0e3));

    }

    return prop;
}



