
#include "../include/utils.cuh"

void VLMO_record_start (hipEvent_t& event_start, hipEvent_t& event_end) {

    cudaErrChk(hipEventCreate(&event_start));
    cudaErrChk(hipEventCreate(&event_end));
    cudaErrChk(hipEventRecord(event_start, NULL));

}


float VLMO_record_end (hipEvent_t& event_start, hipEvent_t& event_end) {

    cudaErrChk(hipEventRecord(event_end, NULL));
    cudaErrChk(hipEventSynchronize(event_end));

    float msec = 0.0f;
    cudaErrChk(hipEventElapsedTime(&msec, event_start, event_end));
    printf("[Perf] Elaped time: %.4f sec\n", msec*1e-3);

    return msec;
}

void get_maximum_size_patch (VLMO_Operator_Descriptor_t& desc) {
    
    
    desc.patch_w = 15;
    desc.patch_h = 15;

}

