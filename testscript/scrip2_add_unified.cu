#include "../include/core.cuh"
#include "../include/operations.cuh"

void test_init (VLMO_Operator_Descriptor_t& desc) {

    srand(0);

    // A
    float* A = (float*) malloc (sizeof (float)*desc.A_h*desc.A_w);
    for (int i=0; i<desc.A_h; i++)
        for (int j=0; j<desc.A_w; j++)
            A[i*desc.A_w+j] = (rand ()%1000-500)/100;
    desc.host_A = A;

    // B
    float* B = (float*) malloc (sizeof (float)*desc.B_h*desc.B_w);
    for (int i=0; i<desc.B_h; i++)
        for (int j=0; j<desc.B_w; j++)
            B[i*desc.B_w+j] = (rand ()%1000-500)/100;
    desc.host_B = B;

    // C
    float* C = (float*) calloc (desc.C_h*desc.C_w, sizeof (float));
    desc.host_C = C;

    size_t total_size = sizeof(float)*desc.A_h*desc.A_w + sizeof(float)*desc.B_h*desc.B_w + sizeof(float)*desc.C_h*desc.C_w;
    printf("[Mem] Host memory allocation completed..\n");
    printf("    total usage usage : %.3f GB\n", total_size*1e-9);

}

void test_result (VLMO_Operator_Descriptor_t& desc) {
    printf("[Test] Start result test..\n");
    for (int i=0; i<desc.C_h; i++)
        for (int j=0; j<desc.C_w; j++) {
            if (desc.device_C[i*desc.C_w+j] != desc.device_A[i*desc.C_w+j] + desc.device_B[i*desc.C_w+j]) {
                printf("[Test] Test failed... C[%d, %d] = %f, but %f\n", i, j, desc.device_A[i*desc.C_w+j] + desc.device_B[i*desc.C_w+j], desc.device_C[i*desc.C_w+j]);
                return ;
            }
        }

    printf("[Test] Test success!\n");
}

int main(void) {


    /****
      *** Very Large Matrices Addition Example with a Single Device
      *** There Matrices stored in memory with "row" major
      ****/

    // Define this problem 
    size_t m = 1024*25;
    size_t n = 1024*25;
    size_t k = 1024*25;
    VLMO_Operator_t op = VLMO_Op_Add_t;
    int device_id = 0;
    

    // Get device information
    size_t free, total;
    hipDeviceProp_t prop =  VLMO_get_device_properties (device_id, &free, &total, true);


    // Make matrix operation description
    VLMO_Operator_Descriptor_t desc;
    desc.op = op;
    desc.A_w = desc.B_h = k;
    desc.B_w = desc.C_w = n;
    desc.A_h = desc.C_h = m;
    desc.flag_unified_mem = true;
    desc.mem_free_size = free;
    desc.num_device = 1;
    desc.prop = prop;

    // Initiate data for test
    test_init (desc);

    // Allocate device memory
    VLMO_malloc_device_mem (desc, true);

    // Launch matrix addtion kernel
    printf("[Func] Addition start..\n");
    VLMO_addition (desc, true);
    
    // Test result
    test_result(desc);

    // Free all memory allocations
    VLMO_clear_all (desc);
    printf("\nEnd..\n");
    return 0;
}

