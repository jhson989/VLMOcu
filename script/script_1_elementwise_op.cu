#include "../include/core.cuh"
#include "../include/operations.cuh"

void test_init (VLMO_Operator_Descriptor_t& desc) {

    srand(0);
    // A
    float* A = (float*) malloc (sizeof (float)*desc.A_h*desc.A_w);
    for (int i=0; i<desc.A_h; i++)
        for (int j=0; j<desc.A_w; j++)
            A[i*desc.A_w+j] = (rand ()%1000-500)/100;
    desc.host_A = A;

    // B
    float* B = (float*) malloc (sizeof (float)*desc.B_h*desc.B_w);

    for (int i=0; i<desc.B_h; i++)
        for (int j=0; j<desc.B_w; j++)
            B[i*desc.B_w+j] = (rand ()%1000-500)/100;
    desc.host_B = B;

    // C
    float* C = (float*) calloc (desc.C_h*desc.C_w, sizeof (float));
    desc.host_C = C;

    size_t total_size = sizeof(float)*desc.A_h*desc.A_w + sizeof(float)*desc.B_h*desc.B_w + sizeof(float)*desc.C_h*desc.C_w;
    printf("[Mem] Host memory allocation completed..\n");
    printf("    total memory  usage : %.3f GB\n", total_size*1e-9);

}

void test_result (VLMO_Operator_Descriptor_t& desc, float* A, float*B, float* C) {

    printf("[Test] Start checking result ..\n");
    float result = 0.0f;
    for (int i=0; i<desc.C_h; i++)
        for (int j=0; j<desc.C_w; j++) {

            switch (desc.op) {
                case VLMO_Op_Element_Add:
                    result = A[i*desc.C_w+j] + B[i*desc.C_w+j];
                    break;
                case VLMO_Op_Element_Sub:
                    result = A[i*desc.C_w+j] - B[i*desc.C_w+j];
                    break;
                case VLMO_Op_Element_Mul:
                    result = A[i*desc.C_w+j] * B[i*desc.C_w+j];
                    break;
                case VLMO_Op_Element_Div:
                    if (B[i*desc.C_w+j] != 0)
                        result = A[i*desc.C_w+j] / B[i*desc.C_w+j];
                    else
                        result = 0.0f;
                    break;
            }
            
            if (C[i*desc.C_w+j] != result) {
                printf("[Test] Test failed... C[%d, %d] = %f, but %f\n", i, j, result, C[i*desc.C_w+j]);
                return ;
            }
        }

    printf("[Test] Test success!\n");
}

int main(void) {


    /****
      *** Very Large Matrices Element-wise Operation Example with a Single Device
      *** These matrices stored in memory with "row" major
      ****/

    // Define a example problem 
    size_t w = 1024*43+1234;
    size_t h = 1024*31+5678;
    printf("Total size of matrix: %.3f GB\n", sizeof(float)*w*h*3*1e-9);
    const int num_op = 4;
    VLMO_Operator_t list_ops[num_op] = {
        VLMO_Op_Element_Add,
        VLMO_Op_Element_Sub,
        VLMO_Op_Element_Mul,
        VLMO_Op_Element_Div
    };
    int device_id = 0;
    
    // Get environment
    size_t free, total;
    hipDeviceProp_t prop =  VLMO_get_device_properties (device_id, &free, &total, false);


    /** Patch based operations **/
    for (int i=0; i<num_op; i++) { 
        VLMO_Operator_t op = list_ops[i];
        printf("=======================================================\n");
        printf ("[%s]\n", VLMO_Op_Name[op].c_str());
        printf("=======================================================\n");

        // Descript a opearator
        VLMO_Operator_Descriptor_t desc;
        desc.op = op;
        desc.A_w = desc.B_w = desc.C_w = w;
        desc.A_h = desc.B_h = desc.C_h = h;
        desc.prop = prop;
        desc.mem_free_size = free;
        desc.num_threads = dim3(256);
        desc.flag_unified_mem=false;

        // Initiate data for test
        test_init (desc);

        // Allocate device memory
        VLMO_malloc_device_mem (desc, true);

        // Launch matrix addtion kernel
        printf("[Func] %s start..\n", VLMO_Op_Name[op].c_str());
        VLMO_element_operation (desc, true);
        
        // Test result
        test_result(desc, desc.host_A, desc.host_B, desc.host_C);

        // Free all memory allocations
        VLMO_clear_all (desc);
        printf("=======================================================\n\n\n");
    }

    /** Unified-memory based operations **/
    for (int i=0; i<num_op; i++) { 
        VLMO_Operator_t op = list_ops[i];
        printf("=======================================================\n");
        printf ("[%s]\n", VLMO_Op_Name[op].c_str());
        printf("=======================================================\n");

        // Descript a opearator
        VLMO_Operator_Descriptor_t desc;
        desc.op = op;
        desc.A_w = desc.B_w = desc.C_w = w;
        desc.A_h = desc.B_h = desc.C_h = h;
        desc.prop = prop;
        desc.mem_free_size = free;
        desc.num_threads = dim3(256);
        desc.flag_unified_mem=true;

        // Initiate data for test
        test_init (desc);

        // Allocate device memory
        VLMO_malloc_device_mem (desc, true);

        // Launch matrix addtion kernel
        printf("[Func] %s start..\n", VLMO_Op_Name[op].c_str());
        VLMO_element_operation (desc, true);
        
        // Test result
        test_result(desc, desc.device_A[0], desc.device_B[0], desc.device_C[0]);

        // Free all memory allocations
        VLMO_clear_all (desc);
        printf("=======================================================\n\n\n");
    }


    return 0;
}

