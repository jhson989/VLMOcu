#include "../include/core.cuh"
#include "../include/operations.cuh"


void test_init (VLMO_Operator_Descriptor_t& desc) {

    srand(0);

    // A
    float* A = (float*) malloc (sizeof (float)*desc.A_h*desc.A_w);
    for (int i=0; i<desc.A_h; i++) {
        for (int j=0; j<desc.A_w; j++) {
            A[i*desc.A_w+j] = (rand ()%1000-500)/100;
        }
    }
    desc.host_A = A;

    // B
    float* B = (float*) malloc (sizeof (float)*desc.B_h*desc.B_w);
    for (int i=0; i<desc.B_h; i++) {
        for (int j=0; j<desc.B_w; j++) {
            B[i*desc.B_w+j] = (rand ()%1000-500)/100;
        }
    }
    desc.host_B = B;

    // C
    float* C = (float*) calloc (desc.C_h*desc.C_w, sizeof (float));
    desc.host_C = C;

    size_t total_size = sizeof(float)*desc.A_h*desc.A_w + sizeof(float)*desc.B_h*desc.B_w + sizeof(float)*desc.C_h*desc.C_w;
    printf("[Mem] Host memory allocation completed..\n");
    printf("    total usage usage : %.3f GB\n", total_size*1e-9);
}

void test_result (VLMO_Operator_Descriptor_t& desc, float* A, float* B, float* C) {

    printf("[Test] Start checking result ..\n");
    for (int i=0; i<desc.C_h; i++) {
        for (int j=0; j<desc.C_w; j++) {

            float result=0.0f;
            for (int l=0; l<desc.B_h; l++) {
                result += A[i*desc.A_w+l]*B[l*desc.B_w+j];
            }

            if (C[i*desc.C_w+j] != result) {
                printf("[Test] Test failed... C[%d, %d] = %f, but %f\n", i, j, result, C[i*desc.C_w+j]);
                return ;
            }
        }
        printf("[%d/%lu]\n", i, desc.C_h);
    }
    printf("[Test] Test success!\n");
}



int main(void) {


    /****
      *** Very Large Matrix Multiplication Example with a Single Device
      *** Matrices stored in memory with "row" major
      ****/

    // Define this problem 
    bool flag_test = true;
    size_t m = 1024*40+19;
    size_t n = 1024*60+18;
    size_t k = 1024*30+17;

    VLMO_Operator_t op = VLMO_Op_Mat_Mul;
    int device_id = 0;
    

    // Get device information
    size_t free, total;
    hipDeviceProp_t prop =  VLMO_get_device_properties (device_id, &free, &total, false);


    // Make matrix operation description
    
    printf ("Do operation %s\n", VLMO_Op_Name[op].c_str());

    VLMO_Operator_Descriptor_t desc;
    desc.op = op;
    desc.A_h = desc.C_h = m;
    desc.B_w = desc.C_w = n;
    desc.A_w = desc.B_h = k;
    desc.flag_unified_mem = false;
    desc.mem_free_size = free;
    desc.num_device = 1;
    desc.prop = prop;
    desc.num_threads = dim3(16, 16);

    // Initiate data for test
    test_init (desc);

    // Allocate device memory
    VLMO_malloc_device_mem (desc, true);

    // Launch matrix addtion kernel
    printf("[Func] %s start..\n", VLMO_Op_Name[op].c_str());
    VLMO_matrix_multiplication (desc, true);
    
    // Test result
    if (flag_test)
        test_result(desc, desc.host_A, desc.host_B, desc.host_C);

    // Free all memory allocations
    VLMO_clear_all (desc);
    printf("\nEnd..\n\n");

    return 0;
}

